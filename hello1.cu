
#include <hip/hip_runtime.h>
#include <stdio.h>

// This is a kernel
__global__ void helloFromGPU(){
    printf("Hello World from GPU!!!\n");
    return;
}

void helloFromCPU(){
    printf("Hello World from CPU!\n");
    return;
}

int main(){

    // Run kernel on GPU: 1 block x 1 thread
    helloFromGPU<<<1,1>>>();
    
    // Run in CPU
    helloFromCPU();
    
    hipDeviceReset();
    hipDeviceSynchronize();
    return 0;
}
