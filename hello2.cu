
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void helloFromGPU(){
    printf("Hello World from GPU! Block number: %d Thread number: %d\n", blockIdx.x ,threadIdx.x);
    return;
}

int main(){
    
    // Thread hierarchy: <<<blocks,threadsPerBlock>>>
    helloFromGPU<<<2147483648,1>>>();
    
    hipDeviceReset();
    hipDeviceSynchronize();
    return 0;
}
