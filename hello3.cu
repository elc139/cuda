
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(){
    printf("Hello World from GPU! Block number: %d Thread number: %d,%d\n", blockIdx.x ,threadIdx.x, threadIdx.y);
    return;
}

int main(){
    
    dim3 threadsPerBlock(512,512);
    helloFromGPU<<<1,threadsPerBlock>>>();
    
    hipDeviceReset();
    hipDeviceSynchronize();
    return 0;
}
